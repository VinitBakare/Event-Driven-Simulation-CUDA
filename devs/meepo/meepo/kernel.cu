#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"

#include ""
#include "../cuda_be/book.h"
#include "circuit.h"
#include "event.h"
#include <vector>

#define THREADS (512)



#include <stdio.h>


 Gate *dev_gates;
 char *dev_nets;
 int *dev_fanouts;
 int *dev_inputs;
 char *dev_GateTable;
 int *dev_EventTable;
 Event *dev_eventsQ;
 int *dev_gatesQ;


 using namespace std;

void processEventsCUDA(vector<Event> &eventQueue,vector<int> &gateQueue,Circuit * circ,int timestep);
void processGatesCUDA(vector<int> &gateQueue,vector<Event> &eventQueue,Circuit * circ,int timestep);
__global__ void eventKernel(Event* dev_events,char* dev_nets,int *dev_fanouts,char* dev_GateTable,int currentTime,int num_gates,int fanwide,int num_events);
__global__ void cleanGateTable(char * dev_GateTable,int num_gates);
__global__ void simulateGatesKernel(int* gatesQ,Gate* dev_gates,char* dev_nets,char* simRes,int Qsize);

void processGatesCUDA(vector<int> &gateQueue,vector<Event> &eventQueue,Circuit *circ, int timestep)
{
	//cout <<"Hendek "<< endl;
	char* simRes;
	char* dev_simRes;
	simRes = new char[gateQueue.size()];
	for(int i=0;i<gateQueue.size();i++) simRes[i]=-1;
	
	HANDLE_ERROR(
		hipMalloc((void**)&dev_simRes,gateQueue.size()*sizeof(char)));
	HANDLE_ERROR(
		hipMemcpy(dev_simRes,simRes,gateQueue.size()*sizeof(char),hipMemcpyHostToDevice));
	HANDLE_ERROR(
		hipMalloc((void**)&dev_gatesQ,gateQueue.size()*sizeof(int)));
	HANDLE_ERROR(
		hipMemcpy(dev_gatesQ,&gateQueue[0],gateQueue.size()*sizeof(int),hipMemcpyHostToDevice));




	////TODO simulate gate
	simulateGatesKernel<<<gateQueue.size()/THREADS+1,THREADS>>>(dev_gatesQ,dev_gates,dev_nets,dev_simRes,gateQueue.size());

	HANDLE_ERROR(
		hipMemcpy(simRes,dev_simRes,gateQueue.size()*sizeof(char),hipMemcpyDeviceToHost));

	for(int i=0;i<gateQueue.size();i++)
	{
		//TODO simulate gate
		//simRes = simulateGate(gateQueue[i]);
		if(simRes[i]!=-1)
		{
			eventQueue.push_back(Event(circ->gates[gateQueue[i]].out,simRes[i],timestep+circ->gates[gateQueue[i]].delay));
		}
	}
	gateQueue.clear();
	hipFree(dev_simRes);
	hipFree(dev_gatesQ);
}

__global__ void simulateGatesKernel(int* dev_gatesQ,Gate* dev_gates,char* dev_nets,char *dev_simRes,int Qsize)
{
	char newVal;
	int gateID;
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i<Qsize)
	{
		gateID = dev_gatesQ[i];
		switch(dev_gates[gateID].type)
		{
		case AND:
			newVal = dev_nets[dev_gates[gateID].in[0]] & dev_nets[dev_gates[gateID].in[1]];
			break;
		case OR:
			newVal = dev_nets[dev_gates[gateID].in[0]] | dev_nets[dev_gates[gateID].in[1]];
			break;
		case NOT:
			newVal = (dev_nets[dev_gates[gateID].in[0]])?(0):(1);
			break;
		default:
			newVal=-2; // ERROR INDEF
			break;
		}
		if(newVal!=dev_gates[gateID].val)
		{
			dev_gates[gateID].val = newVal;
			dev_simRes[i] = newVal;
		}
		else dev_simRes[i] = -1;
	}
}

 void processEventsCUDA(vector<Event> &eventQueue,vector<int> &gateQueue,Circuit * circ,int timestep)
{
	// cout << "CUDA event processing..." << endl; //DEBUG
	char* gatesToBeScheduled =new char[circ->num_gates];

	cleanGateTable<<<((circ->num_gates)/THREADS)+1,THREADS>>>(dev_GateTable,circ->num_gates);
	
	hipFree(dev_eventsQ);

	HANDLE_ERROR(
		hipMalloc((void**)&dev_eventsQ,eventQueue.size()*sizeof(Event)));
	
	HANDLE_ERROR(
		hipMemcpy(dev_eventsQ,&eventQueue[0],eventQueue.size()*sizeof(Event),hipMemcpyHostToDevice));


	eventKernel<<<(eventQueue.size()/32)+1,32>>>(dev_eventsQ,dev_nets,dev_fanouts,dev_GateTable,
											 timestep,circ->num_gates,circ->fanwide,eventQueue.size());

	
	HANDLE_ERROR(
		hipMemcpy(gatesToBeScheduled,dev_GateTable,(circ->num_gates)*sizeof(char),hipMemcpyDeviceToHost));

	HANDLE_ERROR(
		hipMemcpy(circ->nets,dev_nets,(circ->num_nets)*sizeof(char),hipMemcpyDeviceToHost));

	
	//TODO pushback to GATEqueue
	for(int i=0;i<circ->num_gates;i++)
	{
		if(gatesToBeScheduled[i]==1)
			gateQueue.push_back(i);
	}

	while(1)
	{
		int dex=0;
		for( ;dex<eventQueue.size();dex++)
		{
			if(eventQueue[dex].time==timestep)
			{
					eventQueue.erase(eventQueue.begin()+dex);
					break;
			}
		}
		if(dex==eventQueue.size()) break;
	}

	
	//hipFree(dev_events);
	//hipDeviceSynchronize();
}

__global__ void cleanGateTable(char *gateTable,int num_gates)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if(x<num_gates)
		gateTable[x]=0;

}

__global__ void eventKernel(Event* dev_events,char* dev_nets,int *dev_fanouts,char* dev_GateTable,int currentTime,int num_gates,int fanwide,int num_events)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i<num_events)
	{
		if(dev_events[i].time == currentTime)
		{
			dev_nets[dev_events[i].netID] = dev_events[i].value;

			for(int k=0;k<fanwide;k++)
			{
				if( dev_fanouts[(dev_events[i].netID)*fanwide+k ] ==-1) break;
				//TODO solve it!
				dev_GateTable[dev_fanouts[(dev_events[i].netID)*fanwide+k]] = 1;
				//dev_GateTable[0]=1;
			}
		}
	}
}

__host__ void initCUDA(Circuit *circ)
{
	HANDLE_ERROR(
		hipMalloc((void**)&dev_gates,sizeof(Gate)*circ->num_gates));
	HANDLE_ERROR(
		hipMemcpy(dev_gates,circ->gates,sizeof(Gate)*circ->num_gates,hipMemcpyHostToDevice));

	HANDLE_ERROR(
		hipMalloc((void**)&dev_nets,sizeof(int)*circ->num_nets));
	HANDLE_ERROR(
		hipMemcpy(dev_nets,circ->nets,circ->num_nets*sizeof(int),hipMemcpyHostToDevice));

	HANDLE_ERROR(
		hipMalloc((void**)&dev_fanouts,sizeof(int)*(circ->num_nets)*(circ->fanwide)));
	HANDLE_ERROR(
		hipMemcpy(dev_fanouts,circ->fanouts,sizeof(int)*(circ->num_nets)*(circ->fanwide),hipMemcpyHostToDevice));

	HANDLE_ERROR(
		hipMalloc((void**)&dev_GateTable,(circ->num_gates)*sizeof(char)));


	//HANDLE_ERROR(
	//	hipMalloc((void**)&dev_events,10*sizeof(Event)));
	//HANDLE_ERROR(
	//	hipMalloc((void**)&dev_gatesQ,sizeof(int)*circ->num_gates));
	
}



__host__ void cleanCUDA()
{
	hipFree(dev_gates);
	hipFree(dev_eventsQ);
	hipFree(dev_nets);
	hipFree(dev_fanouts);
	hipFree(dev_GateTable);
	hipFree(dev_gatesQ);
}


